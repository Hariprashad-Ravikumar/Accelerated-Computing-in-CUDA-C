
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Initialize array values on the host.
 */

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

/*
 * Double elements in parallel on the GPU.
 */

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

/*
 * Check all elements have been doubled on the host.
 */

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 1000;
  int *a;

  size_t size = N * sizeof(int);

  /*
   * Use `cudaMallocManaged` to allocate pointer `a` available
   * on both the host and the device.
   */

  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 256;
  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  /*
   * Use `cudaFree` to free memory allocated
   * with `cudaMallocManaged`.
   */

  hipFree(a);
}
