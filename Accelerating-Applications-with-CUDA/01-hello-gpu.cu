
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

__global__ void helloGPU()
{
  printf("Hello also from the CPU.\n");
}

int main()
{


  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */

  helloGPU<<<1, 1>>>();
  hipDeviceSynchronize();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
  helloCPU();
  helloGPU<<<1, 1>>>();
  hipDeviceSynchronize();
}
